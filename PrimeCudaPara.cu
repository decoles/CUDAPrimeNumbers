
////////////////////////////////////////////////////////////
//FIND TOTAL PRIME FACTORS IN A RANGE FROM 1 TO N USING CUDA
//CREATED BY DAVID COLES
////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////
//https://docs.nvidia.com/cuda/index.html SOME CODE MAY BE DERIVED FROM SNIPPETS OF STARTER CODE
//FROM OFFICAL NVIDA DOCUMENTATION
//
// https://mae.ufl.edu/~uhk/QUICK-SEMI-PRIME-FACTORING.pdf for some equations help
//http://compoasso.free.fr/primelistweb/page/prime/liste_online_en.php for comfirming largest prime number
////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h> //FOR FILES
#include <time.h>
//NO CUDA DEPENDENCIES REQUIRED AS WE ARE RUNNING WITH NVCC

//preloads array with all 1's
__global__
void setup(int n, unsigned char *isprime) //Global means it will run with the gpu only Cuts 1 second
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; //Will be each thread in each block
    int stride = blockDim.x * gridDim.x; //Will be equivalent to number of blocks , achieves maximum occupancy on device
    for(int i= index +2; i< n; i+= stride) 
    {  
        isprime[i]=1; //Sets isprime values in the whole array to one except first two
    }
}

//Gets a sum of all Prime numbers within range of 1 billion
__global__
void count(int n, unsigned char *isprime, unsigned int *cnt) //Perform Reduction
{
    int local = 0; //PER THREAD variable
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i= index; i< n; i+= stride) 
    {  
        if(isprime[i]) 
        { 
            local++; //Increment is local to each thread, will get each threads sum
        }
    }
    atomicAdd(cnt, local); //atomicAdd is a function that adds the local value to the global value
    //atomic add works similar to mpi reduce where it takes all the values created and adds them together in one spot
}

//HOST FUNCTION, RUNS ON GPU, since no __global__ specificed it defaults to host.
//Control Number from Wolfram states 50,847,534 is correct number
//Could have __host__ device is other option but can only be called by GPU
int main(void)
{
    double fstart, fstop;
    struct timespec start, stop;
    clock_gettime(CLOCK_MONOTONIC, &start); fstart=(double)start.tv_sec + ((double)start.tv_nsec/1000000000.0);

    int N = 1000000000; // 1 billion
    int  j; //Incrmentor for nested loop
    unsigned int p = 2, cnt = 0; //p is initalizer of the main finder loop
    unsigned char *isprime; //Primary char array that controls the program, has 0 or 1 to signify pnum
    unsigned int *dev_cnt;

    //Will require transfering to GPU to use.
    hipMalloc((void**)&dev_cnt, sizeof(unsigned int)); //CUDA MALLOC CANNOT ACCESS GPU CODE HERE
    //Cudamaloc required as CPU dosent handle CudaMallocManaged 
    hipMallocManaged(&isprime,  N * sizeof(unsigned char)); //CPU and GPU accesible
    isprime[0] = 0;
    isprime[1] = 0;
    // Perform setup on GPU with 256 threads
    setup<<<1 , 1024>>>(N, isprime);
    //Kuda kernal Grid size, followed by block size, in this case
    //each block has 1024 threads

    hipDeviceSynchronize(); //Synchronize the GPU, is needed or Segfault will occur
    //sieve-of-eratosthenes algorithm
    while( (p*p) <=  N) //Cant eaisly paralize becuase its mutualy dependant on the whole array
    {
        //All prime numbers but 2 will be odd
        for(j=2*p; j<N+1; j+=p) 
            isprime[j]=0; // Zero is false turns factors of 2 to false basically deletes it

        // find next lowest prime
        for(j=p+1; j<N+1; j++) //Prime number has to have one other number and 1
        { 
            if(isprime[j]) //if value is one turn p to j not a multiple of 2
            {
                p=j; 
                break; //Exit nested looop
            } 
        }
    }
    for(int i = N; i > 0 ; i--) //Look for largests prime that has been found
    {
        if(isprime[i]) //if prime is true (1) then print the index(the  prime number)
        
        {
            printf("The largest prime to a billion is %d\n", i);
            break; //Largest prime has been found no reason to continue
        }
    }
    hipMemcpy(dev_cnt, &cnt, sizeof(unsigned int), hipMemcpyHostToDevice); //Copy to GPU
    //Increasing threads makes proccessed value to small and creates zero
    count<<<4096, 1024>>>(N, isprime, dev_cnt);
    hipMemcpy(&cnt, dev_cnt, sizeof(unsigned int), hipMemcpyDeviceToHost); //Copy from GPU
    printf("Total primes from 0 to a billion %d\n", cnt);
    clock_gettime(CLOCK_MONOTONIC, &stop); fstop=(double)stop.tv_sec + ((double)stop.tv_nsec/1000000000.0);
    printf("completed in %lf seconds\n", (fstop-fstart));

    hipFree(isprime);
    hipFree(dev_cnt); //Frees cuda memory
    return 0;
}