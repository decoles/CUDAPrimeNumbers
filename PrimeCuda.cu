
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>  
#include <time.h>
#define MAX (1000000000)
//THIS RUNS SEQUENTIAL ON THE CPU
unsigned char isprime[MAX+1]; //sets isprime array with a billion values
//unsigned int value[MAX+1];  
//Value does not serve any known purpose. Results do not change when it is removed
//
int main(void)
{
    const int N = 1000000000;
    double fstart, fstop;
    struct timespec start, stop;
    clock_gettime(CLOCK_MONOTONIC, &start); fstart=(double)start.tv_sec + ((double)start.tv_nsec/1000000000.0);

    // unsigned char *isprime;
    // malloc(&isprime,  N * sizeof(unsigned char));
    int i, j;
    unsigned int p=2, cnt=0;
    // not prime by definition
    isprime[0]=0; 
    isprime[1]=0; 
    //Itertes through a billion
    for(i=2; i<MAX+1; i++) 
    {  
        isprime[i]=1; //Sets isprime values in the whole array to one
        //sets whole array to true
       
    }
    //Each time the while roop iterates and visits the for loop  
    //Will increases 
    //p starts as 2 will be 4 initally

    while( (p*p) <=  MAX) 
    {
        // invalidate all multiples of lowest prime so far
        //Essentially removes any non prime numbers
        // Will run over and over, finds prime numbers 
        //Loops are always checking if a prime number or not and will elimniate 
        //when found not to be prime
        //This proccess is much quicker than the other ways of doing things.
        for(j=2*p; j<MAX+1; j+=p) 
            isprime[j]=0; // Zero is false turns factors of 2 to false

        // find next lowest prime
        for(j=p+1; j<MAX+1; j++) 
        { 
            if(isprime[j]) 
            {
                p=j; 
                break; 
            } 
        }
    }
    for(i=0; i<MAX+1; i++) { if(isprime[i]) { cnt++; } }  //Counter, gets a count when isprime is true
    for(i = N; i > 0 ; i--) //Look for largests prime that has been found
    {
        if(isprime[i]) 
        {
            printf("The largest prime to a billion is %d\n", i);
            break;//Largest prime has been found no reason to continue
        }
    }
    //Above scans isPrime array and if i  [i] is true it will increment
    //count, giving us our total number of prime numbers
    printf("\nNumber of primes [0..%d]=%u\n\n", MAX, cnt);
    //Gets end time.
    clock_gettime(CLOCK_MONOTONIC, &stop); fstop=(double)stop.tv_sec + ((double)stop.tv_nsec/1000000000.0);
    printf("completed in %lf seconds\n", (fstop-fstart));
    return 0;
}